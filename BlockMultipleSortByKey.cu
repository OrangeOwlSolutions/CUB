#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <stdio.h>
#include <stdlib.h>

#include "Utilities.cuh"

using namespace hipcub;

/*******************************/
/* CUB BLOCKSORT KERNEL SHARED */
/*******************************/
template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void shared_BlockSortKernel(float *d_valuesA, float *d_valuesB, int *d_keys, float *d_values_resultA, float *d_values_resultB, int *d_keys_result)
{
    // --- Shared memory allocation
	__shared__ float sharedMemoryArrayValuesA[BLOCK_THREADS * ITEMS_PER_THREAD];
	__shared__ float sharedMemoryArrayValuesB[BLOCK_THREADS * ITEMS_PER_THREAD];
	__shared__ int   sharedMemoryArrayKeys[BLOCK_THREADS * ITEMS_PER_THREAD];
	__shared__ int   sharedMemoryHelperIndices[BLOCK_THREADS * ITEMS_PER_THREAD];

	// --- Specialize BlockStore and BlockRadixSort collective types
	typedef hipcub::BlockRadixSort	<int , BLOCK_THREADS, ITEMS_PER_THREAD, int>	BlockRadixSortT;
    
	// --- Allocate type-safe, repurposable shared memory for collectives
    __shared__ typename BlockRadixSortT::TempStorage temp_storage;

	int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);

	// --- Load data to shared memory
    for (int k = 0; k < ITEMS_PER_THREAD; k++) {
		sharedMemoryArrayValuesA [threadIdx.x * ITEMS_PER_THREAD + k] = d_valuesA[block_offset + threadIdx.x * ITEMS_PER_THREAD + k];
		sharedMemoryArrayValuesB [threadIdx.x * ITEMS_PER_THREAD + k] = d_valuesB[block_offset + threadIdx.x * ITEMS_PER_THREAD + k];
		sharedMemoryArrayKeys    [threadIdx.x * ITEMS_PER_THREAD + k] = d_keys   [block_offset + threadIdx.x * ITEMS_PER_THREAD + k];
		sharedMemoryHelperIndices[threadIdx.x * ITEMS_PER_THREAD + k] =                          threadIdx.x * ITEMS_PER_THREAD + k ;
	}
	__syncthreads();

    // --- Collectively sort the keys
    BlockRadixSortT(temp_storage).SortBlockedToStriped(*static_cast<int(*)[ITEMS_PER_THREAD]>(static_cast<void*>(sharedMemoryArrayKeys     + (threadIdx.x * ITEMS_PER_THREAD))),
		                                               *static_cast<int(*)[ITEMS_PER_THREAD]>(static_cast<void*>(sharedMemoryHelperIndices + (threadIdx.x * ITEMS_PER_THREAD))));
    __syncthreads();

	// --- Write data to shared memory
    for (int k = 0; k < ITEMS_PER_THREAD; k++) {
		d_values_resultA[block_offset + threadIdx.x * ITEMS_PER_THREAD + k] = sharedMemoryArrayValuesA[sharedMemoryHelperIndices[threadIdx.x * ITEMS_PER_THREAD + k]];
		d_values_resultB[block_offset + threadIdx.x * ITEMS_PER_THREAD + k] = sharedMemoryArrayValuesB[sharedMemoryHelperIndices[threadIdx.x * ITEMS_PER_THREAD + k]];
		d_keys_result   [block_offset + threadIdx.x * ITEMS_PER_THREAD + k] = sharedMemoryArrayKeys                             [threadIdx.x * ITEMS_PER_THREAD + k];
	}
}

/********/
/* MAIN */
/********/
int main() {

	const int numElemsPerArray  = 8;
	const int numArrays			= 4;
	const int N					= numArrays * numElemsPerArray;
	const int numElemsPerThread = 4;

	const int RANGE				= N * numElemsPerThread;

	// --- Allocating and initializing the data on the host
	float *h_valuesA	= (float *)malloc(N * sizeof(float));
	float *h_valuesB	= (float *)malloc(N * sizeof(float));
	int *h_keys			= (int *)  malloc(N * sizeof(int));
    for (int i = 0 ; i < N; i++) {
		h_valuesA[i] = rand() % RANGE;
		h_valuesB[i] = rand() % RANGE;
		h_keys[i]	 = rand() % RANGE;
	}

	printf("Original\n\n");
	for (int k = 0; k < numArrays; k++) 
		for (int i = 0; i < numElemsPerArray; i++)
			printf("Array nr. %i; Element nr. %i; Key %i; Value A %f; Value B %f\n", k, i, h_keys[k * numElemsPerArray + i], h_valuesA[k * numElemsPerArray + i], h_valuesB[k * numElemsPerArray + i]);

	// --- Allocating the results on the host
	float *h_values_resultA  = (float *)malloc(N * sizeof(float));
	float *h_values_resultB  = (float *)malloc(N * sizeof(float));
	float *h_values_result2  = (float *)malloc(N * sizeof(float));
	int   *h_keys_result1	 = (int *)  malloc(N * sizeof(int));
	int   *h_keys_result2	 = (int *)  malloc(N * sizeof(int));
        
    // --- Allocating space for data and results on device
    float *d_valuesA;			gpuErrchk(hipMalloc((void **)&d_valuesA,		 N * sizeof(float)));
    float *d_valuesB;			gpuErrchk(hipMalloc((void **)&d_valuesB,		 N * sizeof(float)));
    int   *d_keys;				gpuErrchk(hipMalloc((void **)&d_keys,			 N * sizeof(int)));
    float *d_values_resultA;	gpuErrchk(hipMalloc((void **)&d_values_resultA, N * sizeof(float)));
    float *d_values_resultB;	gpuErrchk(hipMalloc((void **)&d_values_resultB, N * sizeof(float)));
    float *d_values_result2;	gpuErrchk(hipMalloc((void **)&d_values_result2, N * sizeof(float)));
    int   *d_keys_result1;		gpuErrchk(hipMalloc((void **)&d_keys_result1,   N * sizeof(int)));
    int   *d_keys_result2;		gpuErrchk(hipMalloc((void **)&d_keys_result2,   N * sizeof(int)));

	// --- BlockSortKernel with shared
	gpuErrchk(hipMemcpy(d_valuesA, h_valuesA, N * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_valuesB, h_valuesB, N * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_keys,   h_keys,   N * sizeof(int),   hipMemcpyHostToDevice));
    shared_BlockSortKernel<N / numArrays / numElemsPerThread, numElemsPerThread><<<numArrays, numElemsPerArray / numElemsPerThread>>>(d_valuesA, d_valuesB, d_keys, d_values_resultA, d_values_resultB, d_keys_result1); 
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());    
	gpuErrchk(hipMemcpy(h_values_resultA, d_values_resultA, N * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_values_resultB, d_values_resultB, N * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_keys_result1,   d_keys_result1,   N * sizeof(int),   hipMemcpyDeviceToHost));
    
	printf("\n\nBlockSortKernel using shared memory\n\n");
	for (int k = 0; k < numArrays; k++) 
		for (int i = 0; i < numElemsPerArray; i++)
			printf("Array nr. %i; Element nr. %i; Key %i; Value %f; Value %f\n", k, i, h_keys_result1[k * numElemsPerArray + i], h_values_resultA[k * numElemsPerArray + i], h_values_resultB[k * numElemsPerArray + i]);

	return 0;
}
