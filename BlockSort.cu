#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <stdio.h>
#include <stdlib.h>

#include "Utilities.cuh"

using namespace hipcub;

/**********************************/
/* CUB BLOCKSORT KERNEL NO SHARED */
/**********************************/
template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void BlockSortKernel(int *d_in, int *d_out)
{
	// --- Specialize BlockLoad, BlockStore, and BlockRadixSort collective types
	typedef hipcub::BlockLoad		<int*, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_TRANSPOSE>	BlockLoadT;
	typedef hipcub::BlockStore		<int*, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_STORE_TRANSPOSE>	BlockStoreT;
	typedef hipcub::BlockRadixSort	<int , BLOCK_THREADS, ITEMS_PER_THREAD>							BlockRadixSortT;

	// --- Allocate type-safe, repurposable shared memory for collectives
	__shared__ union {
		typename BlockLoadT		::TempStorage load;
		typename BlockStoreT	::TempStorage store;
		typename BlockRadixSortT::TempStorage sort;
    } temp_storage;

	// --- Obtain this block's segment of consecutive keys (blocked across threads)
	int thread_keys[ITEMS_PER_THREAD];
	int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);

	BlockLoadT(temp_storage.load).Load(d_in + block_offset, thread_keys);
	__syncthreads(); 

	// --- Collectively sort the keys
	BlockRadixSortT(temp_storage.sort).Sort(thread_keys);
	__syncthreads(); 

	// --- Store the sorted segment
	BlockStoreT(temp_storage.store).Store(d_out + block_offset, thread_keys);
  
}

/*******************************/
/* CUB BLOCKSORT KERNEL SHARED */
/*******************************/
template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void shared_BlockSortKernel(int *d_in, int *d_out)
{
    // --- Shared memory allocation
	__shared__ int sharedMemoryArray[BLOCK_THREADS * ITEMS_PER_THREAD];

	// --- Specialize BlockStore and BlockRadixSort collective types
	typedef hipcub::BlockRadixSort	<int , BLOCK_THREADS, ITEMS_PER_THREAD>	BlockRadixSortT;
    
	// --- Allocate type-safe, repurposable shared memory for collectives
    __shared__ typename BlockRadixSortT::TempStorage temp_storage;

	int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);

	// --- Load data to shared memory
    for (int k = 0; k < ITEMS_PER_THREAD; k++) sharedMemoryArray[threadIdx.x * ITEMS_PER_THREAD + k]  = d_in[block_offset + threadIdx.x * ITEMS_PER_THREAD + k];
	__syncthreads();

    // --- Collectively sort the keys
    BlockRadixSortT(temp_storage).Sort(*static_cast<int(*)[ITEMS_PER_THREAD]>(static_cast<void*>(sharedMemoryArray + (threadIdx.x * ITEMS_PER_THREAD))));
    __syncthreads();

	// --- Write data to shared memory
    for (int k = 0; k < ITEMS_PER_THREAD; k++) d_out[block_offset + threadIdx.x * ITEMS_PER_THREAD + k] = sharedMemoryArray[threadIdx.x * ITEMS_PER_THREAD + k];
	
}

/********/
/* MAIN */
/********/
int main() {

	const int numElemsPerArray  = 8;
	const int numArrays			= 4;
	const int N					= numArrays * numElemsPerArray;
	const int numElemsPerThread = 4;

	const int RANGE				= N * numElemsPerThread;

	// --- Allocating and initializing the data on the host
	int *h_data	= (int *)malloc(N * sizeof(int));
    for (int i = 0 ; i < N; i++) h_data[i] = rand() % RANGE;

	// --- Allocating the results on the host
	int *h_result1 = (int *)malloc(N * sizeof(int));
	int *h_result2 = (int *)malloc(N * sizeof(int));
        
    // --- Allocating space for data and results on device
    int *d_in;		gpuErrchk(hipMalloc((void **)&d_in,   N * sizeof(int)));
    int *d_out1;	gpuErrchk(hipMalloc((void **)&d_out1, N * sizeof(int)));
    int *d_out2;	gpuErrchk(hipMalloc((void **)&d_out2, N * sizeof(int)));

	// --- BlockSortKernel no shared
	gpuErrchk(hipMemcpy(d_in, h_data, N*sizeof(int), hipMemcpyHostToDevice));
	BlockSortKernel<N / numArrays / numElemsPerThread, numElemsPerThread><<<numArrays, numElemsPerArray / numElemsPerThread>>>(d_in, d_out1); 
    gpuErrchk(hipMemcpy(h_result1, d_out1, N*sizeof(int), hipMemcpyDeviceToHost));
    
	printf("BlockSortKernel no shared\n\n");
	for (int k = 0; k < numArrays; k++) 
		for (int i = 0; i < numElemsPerArray; i++)
			printf("Array nr. %i; Element nr. %i; Value %i\n", k, i, h_result1[k * numElemsPerArray + i]);
	
	// --- BlockSortKernel with shared
	gpuErrchk(hipMemcpy(d_in, h_data, N*sizeof(int), hipMemcpyHostToDevice));
    shared_BlockSortKernel<N / numArrays / numElemsPerThread, numElemsPerThread><<<numArrays, numElemsPerArray / numElemsPerThread>>>(d_in, d_out2); 
    gpuErrchk(hipMemcpy(h_result2, d_out2, N*sizeof(int), hipMemcpyDeviceToHost));
    
	printf("\n\nBlockSortKernel with shared\n\n");
	for (int k = 0; k < numArrays; k++) 
		for (int i = 0; i < numElemsPerArray; i++)
			printf("Array nr. %i; Element nr. %i; Value %i\n", k, i, h_result2[k * numElemsPerArray + i]);

	return 0;
}
