#define HIPCUB_STDERR

#include <stdio.h>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include <cub/device/device_reduce.cuh>

#include "TimingGPU.cuh"
#include "Utilities.cuh"

using namespace hipcub;

/********/
/* MAIN */
/********/
int main() {

    const int N = 8388608;

    gpuErrchk(hipFree(0));

    float *h_data		= (float *)malloc(N * sizeof(float));
	float h_result = 0.f;

	for (int i=0; i<N; i++) {
		h_data[i] = 3.f;
		h_result = h_result + h_data[i];
	}

	TimingGPU timerGPU;

	float *d_data;			gpuErrchk(hipMalloc((void**)&d_data, N * sizeof(float)));
    gpuErrchk(hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice));

    /**********/
    /* THRUST */
    /**********/
    timerGPU.StartCounter();
    thrust::device_ptr<float> wrapped_ptr = thrust::device_pointer_cast(d_data);
	float h_result1 = thrust::reduce(wrapped_ptr, wrapped_ptr + N);
	printf("Timing for Thrust = %f\n", timerGPU.GetCounter());

    /*******/
    /* CUB */
    /*******/
    timerGPU.StartCounter();
    float			*h_result2 = (float *)malloc(sizeof(float));
    float			*d_result2;	gpuErrchk(hipMalloc((void**)&d_result2, sizeof(float)));
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;

	DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_data, d_result2, N);
	gpuErrchk(hipMalloc((void**)&d_temp_storage, temp_storage_bytes));
	DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_data, d_result2, N);

	gpuErrchk(hipMemcpy(h_result2, d_result2, sizeof(float), hipMemcpyDeviceToHost));

	printf("Timing for CUB = %f\n", timerGPU.GetCounter());

	printf("Results:\n");
	printf("Exact: %f\n", h_result);
	printf("Thrust: %f\n", h_result1);
	printf("CUB: %f\n", h_result2[0]);

}
