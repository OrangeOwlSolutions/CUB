#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <stdio.h>
#include <stdlib.h>

#include "Utilities.cuh"

using namespace hipcub;

/**********************************/
/* CUB BLOCKSORT KERNEL NO SHARED */
/**********************************/
template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void BlockSortKernel(float *d_values, int *d_keys, float *d_values_result, int *d_keys_result)
{
	// --- Specialize BlockLoad, BlockStore, and BlockRadixSort collective types
	typedef hipcub::BlockLoad		<int*,   BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_TRANSPOSE>	 BlockLoadIntT;
	typedef hipcub::BlockLoad		<float*, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_TRANSPOSE>	 BlockLoadFloatT;
	typedef hipcub::BlockStore		<int*,   BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_STORE_TRANSPOSE> BlockStoreIntT;
	typedef hipcub::BlockStore		<float*, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_STORE_TRANSPOSE> BlockStoreFloatT;
	typedef hipcub::BlockRadixSort	<int ,   BLOCK_THREADS, ITEMS_PER_THREAD, float>				 BlockRadixSortT;

	// --- Allocate type-safe, repurposable shared memory for collectives
	__shared__ union {
		typename BlockLoadIntT		::TempStorage loadInt;
		typename BlockLoadFloatT	::TempStorage loadFloat;
		typename BlockStoreIntT		::TempStorage storeInt;
		typename BlockStoreFloatT	::TempStorage storeFloat;
		typename BlockRadixSortT	::TempStorage sort;
    } temp_storage;

	// --- Obtain this block's segment of consecutive keys (blocked across threads)
	int   thread_keys[ITEMS_PER_THREAD];
	float thread_values[ITEMS_PER_THREAD];
	int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);

	BlockLoadIntT(temp_storage.loadInt).Load(d_keys   + block_offset, thread_keys);
	BlockLoadFloatT(temp_storage.loadFloat).Load(d_values + block_offset, thread_values);
	__syncthreads(); 

	// --- Collectively sort the keys
	BlockRadixSortT(temp_storage.sort).SortBlockedToStriped(thread_keys, thread_values);
	__syncthreads(); 

	// --- Store the sorted segment
	BlockStoreIntT(temp_storage.storeInt).Store(d_keys_result   + block_offset, thread_keys);
	BlockStoreFloatT(temp_storage.storeFloat).Store(d_values_result + block_offset, thread_values);
  
}

/*******************************/
/* CUB BLOCKSORT KERNEL SHARED */
/*******************************/
template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void shared_BlockSortKernel(float *d_values, int *d_keys, float *d_values_result, int *d_keys_result)
{
    // --- Shared memory allocation
	__shared__ float sharedMemoryArrayValues[BLOCK_THREADS * ITEMS_PER_THREAD];
	__shared__ int   sharedMemoryArrayKeys[BLOCK_THREADS * ITEMS_PER_THREAD];

	// --- Specialize BlockStore and BlockRadixSort collective types
	typedef hipcub::BlockRadixSort	<int , BLOCK_THREADS, ITEMS_PER_THREAD, float>	BlockRadixSortT;
    
	// --- Allocate type-safe, repurposable shared memory for collectives
    __shared__ typename BlockRadixSortT::TempStorage temp_storage;

	int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);

	// --- Load data to shared memory
    for (int k = 0; k < ITEMS_PER_THREAD; k++) {
		sharedMemoryArrayValues[threadIdx.x * ITEMS_PER_THREAD + k] = d_values[block_offset + threadIdx.x * ITEMS_PER_THREAD + k];
		sharedMemoryArrayKeys[threadIdx.x * ITEMS_PER_THREAD + k]   = d_keys[block_offset + threadIdx.x * ITEMS_PER_THREAD + k];
	}
	__syncthreads();

    // --- Collectively sort the keys
    BlockRadixSortT(temp_storage).SortBlockedToStriped(*static_cast<int(*)  [ITEMS_PER_THREAD]>(static_cast<void*>(sharedMemoryArrayKeys   + (threadIdx.x * ITEMS_PER_THREAD))),
		                                               *static_cast<float(*)[ITEMS_PER_THREAD]>(static_cast<void*>(sharedMemoryArrayValues + (threadIdx.x * ITEMS_PER_THREAD))));
    __syncthreads();

	// --- Write data to shared memory
    for (int k = 0; k < ITEMS_PER_THREAD; k++) {
		d_values_result[block_offset + threadIdx.x * ITEMS_PER_THREAD + k] = sharedMemoryArrayValues[threadIdx.x * ITEMS_PER_THREAD + k];
		d_keys_result  [block_offset + threadIdx.x * ITEMS_PER_THREAD + k] = sharedMemoryArrayKeys  [threadIdx.x * ITEMS_PER_THREAD + k];
	}
}

/********/
/* MAIN */
/********/
int main() {

	const int numElemsPerArray  = 8;
	const int numArrays			= 4;
	const int N					= numArrays * numElemsPerArray;
	const int numElemsPerThread = 4;

	const int RANGE				= N * numElemsPerThread;

	// --- Allocating and initializing the data on the host
	float *h_values	= (float *)malloc(N * sizeof(float));
	int *h_keys		= (int *)  malloc(N * sizeof(int));
    for (int i = 0 ; i < N; i++) {
		h_values[i] = rand() % RANGE;
		h_keys[i]	= rand() % RANGE;
	}

	printf("Original\n\n");
	for (int k = 0; k < numArrays; k++) 
		for (int i = 0; i < numElemsPerArray; i++)
			printf("Array nr. %i; Element nr. %i; Key %i; Value %f\n", k, i, h_keys[k * numElemsPerArray + i], h_values[k * numElemsPerArray + i]);

	// --- Allocating the results on the host
	float *h_values_result1 = (float *)malloc(N * sizeof(float));
	float *h_values_result2 = (float *)malloc(N * sizeof(float));
	int   *h_keys_result1	= (int *)  malloc(N * sizeof(int));
	int   *h_keys_result2	= (int *)  malloc(N * sizeof(int));
        
    // --- Allocating space for data and results on device
    float *d_values;			gpuErrchk(hipMalloc((void **)&d_values,		 N * sizeof(float)));
    int   *d_keys;				gpuErrchk(hipMalloc((void **)&d_keys,			 N * sizeof(int)));
    float *d_values_result1;	gpuErrchk(hipMalloc((void **)&d_values_result1, N * sizeof(float)));
    float *d_values_result2;	gpuErrchk(hipMalloc((void **)&d_values_result2, N * sizeof(float)));
    int   *d_keys_result1;		gpuErrchk(hipMalloc((void **)&d_keys_result1,   N * sizeof(int)));
    int   *d_keys_result2;		gpuErrchk(hipMalloc((void **)&d_keys_result2,   N * sizeof(int)));

	// --- BlockSortKernel no shared
	gpuErrchk(hipMemcpy(d_values, h_values, N * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_keys,   h_keys,   N * sizeof(int),   hipMemcpyHostToDevice));
	BlockSortKernel<N / numArrays / numElemsPerThread, numElemsPerThread><<<numArrays, numElemsPerArray / numElemsPerThread>>>(d_values, d_keys, d_values_result1, d_keys_result1); 
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());    
	gpuErrchk(hipMemcpy(h_values_result1, d_values_result1, N * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_keys_result1,   d_keys_result1,   N * sizeof(int),   hipMemcpyDeviceToHost));
    
	printf("\n\nBlockSortKernel no shared\n\n");
	for (int k = 0; k < numArrays; k++) 
		for (int i = 0; i < numElemsPerArray; i++)
			printf("Array nr. %i; Element nr. %i; Key %i; Value %f\n", k, i, h_keys_result1[k * numElemsPerArray + i], h_values_result1[k * numElemsPerArray + i]);
	
	// --- BlockSortKernel with shared
	gpuErrchk(hipMemcpy(d_values, h_values, N * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_keys,   h_keys,   N * sizeof(int),   hipMemcpyHostToDevice));
    shared_BlockSortKernel<N / numArrays / numElemsPerThread, numElemsPerThread><<<numArrays, numElemsPerArray / numElemsPerThread>>>(d_values, d_keys, d_values_result2, d_keys_result2); 
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());    
	gpuErrchk(hipMemcpy(h_values_result2, d_values_result2, N * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_keys_result2,   d_keys_result2,   N * sizeof(int),   hipMemcpyDeviceToHost));
    
	printf("\n\nBlockSortKernel shared\n\n");
	for (int k = 0; k < numArrays; k++) 
		for (int i = 0; i < numElemsPerArray; i++)
			printf("Array nr. %i; Element nr. %i; Key %i; Value %f\n", k, i, h_keys_result2[k * numElemsPerArray + i], h_values_result2[k * numElemsPerArray + i]);

	return 0;
}
